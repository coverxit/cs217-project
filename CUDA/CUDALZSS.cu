#include "hip/hip_runtime.h"
#include <utility>

#include <memory.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "../Settings.h"

#include "../BitHelper.h"
#include "../LZSSInterface.h"
#include "../TimerHelper.hpp"

#include "../MatchHelper/MatchHelper.h"
#include "CUDALZSS.h"

#define cudaCheckError(op, msg) \
    do { hipError_t ret = (op); if ((ret) != hipSuccess) _gerror((ret), (msg)); } while (false)

#define MIN(a, b) \
    ((a) < (b) ? (a) : (b))

inline void _gerror(hipError_t hipError_t, const char* msg) {
    fprintf(stderr, "%s, hipError_t = %d\n", msg, hipError_t);
    exit(-1);
}

__global__ void CompressKernel(const uint8_t* deviceInBuf, int inSize,
    uint8_t* deviceOutBuf, int* deviceOutSize,
    CompressFlagBlock* deviceFlagOut, int nFlagBlocks, int* deviceFlagSize)
{
    __shared__ uint8_t blockBuf[DataBlockSize];
    __shared__ PairType blockFlags[DataBlockSize];
    __shared__ CompressFlagBlock compressBlock;

    auto threadId = threadIdx.x;
    auto blockId = blockIdx.x;

    auto blockOffset = blockIdx.x * DataBlockSize;
    auto blockSize = MIN(DataBlockSize, inSize - blockOffset);

    for (int t = threadId; t < blockSize; t += blockDim.x) {
        blockBuf[t] = deviceInBuf[blockOffset + t];
    }

    for (int t = threadId; t < DataBlockSize / 8; t += blockDim.x) {
        compressBlock.Flags[t] = 0;
    }
    __syncthreads();

    for (int t = threadId; t < blockSize; t += blockDim.x) {
        auto lookbackLength = MIN(WindowSize, t);
        auto lookaheadLength = MIN(MaxEncodeLength, blockSize - t);
        int matchOffset, matchLength;

        if (FindMatch(blockBuf + t - lookbackLength, lookbackLength,
            blockBuf + t, lookaheadLength, matchOffset, matchLength)) {

            // Convert offset to backward representation
            matchOffset = lookbackLength - matchOffset;

            // Due to the bit limit, minus 1 for exact offset and length
            blockFlags[t] = ((matchOffset - 1) << PairLengthBits) | (matchLength - 1);
        } else {
            blockFlags[t] = 0;
        }
    }
    __syncthreads();

    // Collector
    if (threadId == 0) {
        compressBlock.CompressedSize = 0;
        compressBlock.NumOfFlags = 0;
        
        for (int i = 0; i < blockSize; ) {
            if (blockFlags[i] == 0) {
                deviceOutBuf[blockOffset + compressBlock.CompressedSize] = blockBuf[i];
                ++compressBlock.CompressedSize;

                PUT_BIT(compressBlock.Flags, compressBlock.NumOfFlags, 0);
                i += 1;
            } else {
                // Plus 1 for the opposite operation in compression
                auto matchLength = (blockFlags[i] & (MaxEncodeLength - 1)) + 1;

                memcpy(deviceOutBuf + blockOffset + compressBlock.CompressedSize, &blockFlags[i], sizeof(PairType));
                compressBlock.CompressedSize += sizeof(PairType);

                PUT_BIT(compressBlock.Flags, compressBlock.NumOfFlags, 1);
                i += matchLength;
            }

            ++compressBlock.NumOfFlags;
        }

        memcpy(deviceFlagOut + blockId, &compressBlock, sizeof(CompressFlagBlock));

        // taken by current flag block
        atomicAdd(deviceFlagSize, SIZE_OF_FLAGS(compressBlock.NumOfFlags) + sizeof(CompressFlagBlock::NumOfFlags)
            + sizeof(CompressFlagBlock::CompressedSize));
        atomicAdd(deviceOutSize, compressBlock.CompressedSize);

        printf("Block %d/%d done.\n", blockId, nFlagBlocks);
    }
}

std::pair<bool, double> CUDALZSS::compress(const uint8_t* inBuf, int inSize,
    uint8_t* outBuf, int& outSize,
    CompressFlagBlock* flagOut, int nFlagBlocks, int& flagSize)
{
    Timer timer(false);

    uint8_t *deviceInBuf, *deviceOutBuf;
    CompressFlagBlock *deviceFlagOut;
    int *deviceOutSize, *deviceFlagSize;

    // Allocate ----------------------------------
    printf("Allocating device variables...\n");
    cudaCheckError(hipMalloc((void**) &deviceInBuf, inSize), "Failed to allocate deviceInBuf");

    cudaCheckError(hipMalloc((void**) &deviceOutBuf, outSize), "Failed to allocate deviceOutBuf");
    cudaCheckError(hipMalloc((void**) &deviceOutSize, sizeof(int)), "Failed to allocate deviceOutSize");

    cudaCheckError(hipMalloc((void**) &deviceFlagOut, sizeof(CompressFlagBlock) * nFlagBlocks), 
        "Failed to allocate deviceFlagOut");
    cudaCheckError(hipMalloc((void**) &deviceFlagSize, sizeof(int)), "Failed to allocate deviceFlagSize");

    // Copy: host to device -----------------------
    printf("Copying data from host to device...\n");
    cudaCheckError(hipMemcpy(deviceInBuf, inBuf, inSize, hipMemcpyHostToDevice), 
        "Failed to copy deviceInBuf to device");
    cudaCheckError(hipMemset(deviceFlagOut, 0, sizeof(CompressFlagBlock) * nFlagBlocks), 
        "Failed to set deviceFlagOut to 0");
    hipDeviceSynchronize();

    // Launch kernel ------------------------------
    printf("Launching kernel...\n");

    timer.begin();
    CompressKernel<<<nFlagBlocks, GPUBlockSize>>>(deviceInBuf, inSize, 
        deviceOutBuf, deviceOutSize, deviceFlagOut, nFlagBlocks, deviceFlagSize);
    auto elapsed = timer.end();
    cudaCheckError(hipDeviceSynchronize(), "Failed to launch kernel");

    // Copy: device to host -----------------------
    printf("Copying data from device to host...\n");
    cudaCheckError(hipMemcpy(&outSize, deviceOutSize, sizeof(int), hipMemcpyDeviceToHost),
        "Failed to copy deviceOutSize to host");
    cudaCheckError(hipMemcpy(&flagSize, deviceFlagSize, sizeof(int), hipMemcpyDeviceToHost),
        "Failed to copy deviceFlagSize to host");

    cudaCheckError(hipMemcpy(outBuf, deviceOutBuf, outSize, hipMemcpyDeviceToHost), 
        "Failed to copy deviceOutBuf to host");
    cudaCheckError(hipMemcpy(flagOut, deviceFlagOut, sizeof(CompressFlagBlock) * nFlagBlocks, hipMemcpyDeviceToHost),
        "Failed to copy deviceFlagOut to host");
    hipDeviceSynchronize();
    
    hipFree(deviceInBuf);
    hipFree(deviceOutBuf);
    hipFree(deviceOutSize);
    hipFree(deviceFlagOut);
    hipFree(deviceFlagSize);

    return std::make_pair(true, elapsed);
}

std::pair<bool, double> CUDALZSS::decompress(CompressFlagBlock* flagIn, int nFlagBlocks,
    const uint8_t* inBuf, int inSize, uint8_t* outBuf)
{
    Timer timer(false);

    timer.begin();
    //DecompressKernel<<<,>>>();
    auto elapsed = timer.end();

    return std::make_pair(true, elapsed);
}
