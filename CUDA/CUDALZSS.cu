#include "hip/hip_runtime.h"
#include <algorithm>
#include <utility>

#include <memory.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "../Settings.h"

#include "../BitHelper.h"
#include "../LZSSInterface.h"
#include "../TimerHelper.hpp"

#include "../MatchHelper/MatchHelper.h"
#include "CUDALZSS.h"
#include "Kernel.h"

#define cudaCheckError(op, msg)    \
    do {                           \
        hipError_t ret = (op);    \
        if ((ret) != hipSuccess)  \
            _gerror((ret), (msg)); \
    } while (false)

inline void _gerror(hipError_t hipError_t, const char* msg)
{
    fprintf(stderr, "%s, CUDA Error (%d): %s\n", msg, hipError_t, hipGetErrorString(hipError_t));
    exit(-1);
}

std::pair<bool, double> CUDALZSS::compress(const uint8_t* inBuf, int inSize,
    uint8_t* outBuf, int& outSize,
    CompressFlagBlock* flagOut, int nFlagBlocks, int& flagSize)
{
    Timer timer(false);

    uint8_t **deviceInBuf, **deviceOutBuf;
    CompressFlagBlock** deviceFlagOut;
    int **deviceOutSize, **deviceFlagSize;
    int *hostOutSize, *hostFlagSize;

    int numOfGPUs = 0;
    hipGetDeviceCount(&numOfGPUs);

    auto numOfKernels = std::min(nFlagBlocks / numOfGPUs, numOfGPUs);
    auto blocksPerKernel = (nFlagBlocks - 1) / numOfKernels + 1;
    auto alignedKernelSize = blocksPerKernel * DataBlockSize;

    // Allocate ----------------------------------
    printf("Allocating device variables... ");
    timer.begin();

    deviceInBuf = new uint8_t*[numOfKernels];
    deviceOutBuf = new uint8_t*[numOfKernels];
    deviceFlagOut = new CompressFlagBlock*[numOfKernels];
    deviceOutSize = new int*[numOfKernels];
    deviceFlagSize = new int*[numOfKernels];

    hostOutSize = new int[numOfKernels];
    hostFlagSize = new int[numOfKernels];

    for (int i = 0; i < numOfKernels; ++i) {
        cudaCheckError(hipSetDevice(i), "Failed to set device");

        auto kernelSize = std::min(alignedKernelSize, inSize - i * alignedKernelSize);
        auto numOfBlock = std::min(blocksPerKernel, nFlagBlocks - i * blocksPerKernel);

        cudaCheckError(hipMalloc((void**)&deviceInBuf[i], kernelSize), "Failed to allocate deviceInBuf");

        cudaCheckError(hipMalloc((void**)&deviceOutBuf[i], kernelSize), "Failed to allocate deviceOutBuf");
        cudaCheckError(hipMalloc((void**)&deviceOutSize[i], sizeof(int)), "Failed to allocate deviceOutSize");

        cudaCheckError(hipMalloc((void**)&deviceFlagOut[i], sizeof(CompressFlagBlock) * numOfBlock),
            "Failed to allocate deviceFlagOut");
        cudaCheckError(hipMalloc((void**)&deviceFlagSize[i], sizeof(int)), "Failed to allocate deviceFlagSize");

        hipDeviceSynchronize();
    }
    printf("%.6fs\n", timer.end());

    // Copy: host to device -----------------------
    printf("Copying from host to device => GPU");
    timer.begin();

    for (int i = 0; i < numOfKernels; ++i) {
        cudaCheckError(hipSetDevice(i), "Failed to set device");

        auto kernelSize = std::min(alignedKernelSize, inSize - i * alignedKernelSize);
        auto numOfBlock = std::min(blocksPerKernel, nFlagBlocks - i * blocksPerKernel);

        printf(" [%d]", i);
        fflush(stdout);
        
        cudaCheckError(hipMemcpyAsync(deviceInBuf[i],
                           inBuf + i * alignedKernelSize, kernelSize,
                           hipMemcpyHostToDevice),
            "Failed to copy inBuf to device");

        cudaCheckError(hipMemsetAsync(deviceFlagOut[i], 0, sizeof(CompressFlagBlock) * numOfBlock),
            "Failed to set deviceFlagOut to 0");

        cudaCheckError(hipMemsetAsync(deviceOutSize[i], 0, sizeof(int)), "Failed to set deviceOutSize to 0");
        cudaCheckError(hipMemsetAsync(deviceFlagSize[i], 0, sizeof(int)), "Failed to set deviceFlagSize to 0");
    }

    for (int i = 0; i < numOfKernels; ++i) {
        cudaCheckError(hipSetDevice(i), "Failed to set device");
        cudaCheckError(hipDeviceSynchronize(), "Failed to synchronize");
    }
    printf("... %.6fs\n", timer.end());
    
    // Launch kernel ----------------------------------
    printf("Launching kernel => GPU");
    timer.begin();

    for (int i = 0; i < numOfKernels; ++i) {
        cudaCheckError(hipSetDevice(i), "Failed to set device");

        auto kernelSize = std::min(alignedKernelSize, inSize - i * alignedKernelSize);
        auto numOfBlock = std::min(blocksPerKernel, nFlagBlocks - i * blocksPerKernel);

        printf(" [%d]", i);
        fflush(stdout);

        CompressKernel<<<numOfBlock, GPUBlockSize>>>(
            deviceInBuf[i], kernelSize,
            deviceOutBuf[i], deviceOutSize[i],
            deviceFlagOut[i], deviceFlagSize[i]);
    }

    for (int i = 0; i < numOfKernels; ++i) {
        cudaCheckError(hipSetDevice(i), "Failed to set device");
        cudaCheckError(hipDeviceSynchronize(), "Failed to launch multi-GPU kernel");
    }
    auto elasped = timer.end();
    printf("... %.6fs\n", elasped);

    // Copy: device to host ---------------------------
    printf("Copying from device to host => GPU");
    timer.begin();

    for (int i = 0; i < numOfKernels; ++i) {
        cudaCheckError(hipSetDevice(i), "Failed to set device");

        auto kernelSize = std::min(alignedKernelSize, inSize - i * alignedKernelSize);
        auto numOfBlock = std::min(blocksPerKernel, nFlagBlocks - i * blocksPerKernel);

        printf(" [%d]", i);
        fflush(stdout);

        cudaCheckError(hipMemcpyAsync(outBuf + i * alignedKernelSize,
                           deviceOutBuf[i], kernelSize,
                           hipMemcpyDeviceToHost),
            "Failed to copy deviceOutBuf to host");

        cudaCheckError(hipMemcpyAsync(flagOut + i * blocksPerKernel,
                           deviceFlagOut[i], numOfBlock * sizeof(CompressFlagBlock),
                           hipMemcpyDeviceToHost),
            "Failed to copy deviceFlagOut to host");

        cudaCheckError(hipMemcpyAsync(hostOutSize + i, deviceOutSize[i],
                           sizeof(int), hipMemcpyDeviceToHost),
            "Failed to copy deviceOutSize to host");
        cudaCheckError(hipMemcpyAsync(hostFlagSize + i, deviceFlagSize[i],
                           sizeof(int), hipMemcpyDeviceToHost),
            "Failed to copy deviceFlagSize to host");
    }

    for (int i = 0; i < numOfKernels; ++i) {
        cudaCheckError(hipSetDevice(i), "Failed to set device");
        cudaCheckError(hipDeviceSynchronize(), "Failed to synchronize");
    }
    printf("... %.6fs\n", timer.end());

    printf("Post processing and cleanup... ");
    timer.begin();

    // Post process ------------------------------
    outSize = flagSize = 0;
    for (int i = 0; i < numOfKernels; ++i) {
        outSize += hostOutSize[i];
        flagSize += hostFlagSize[i];
    }

    for (int i = 0; i < numOfKernels; ++i) {
        cudaCheckError(hipSetDevice(i), "Failed to set device");
        hipFree(deviceInBuf[i]);
        hipFree(deviceOutBuf[i]);
        hipFree(deviceOutSize[i]);
        hipFree(deviceFlagOut[i]);
        hipFree(deviceFlagSize[i]);
    }

    delete[] deviceInBuf;
    delete[] deviceOutBuf;
    delete[] deviceFlagOut;
    delete[] deviceOutSize;
    delete[] deviceFlagSize;

    delete[] hostOutSize;
    delete[] hostFlagSize;

    printf("%.6fs\n", timer.end());
    return std::make_pair(true, elasped);
}

std::pair<bool, double> CUDALZSS::decompress(CompressFlagBlock* flagIn, int nFlagBlocks,
    const uint8_t* inBuf, int inSize, uint8_t* outBuf, int outSize)
{
    Timer timer(false);

    uint8_t **deviceInBuf, **deviceOutBuf;
    CompressFlagBlock** deviceFlagIn;

    int numOfGPUs = 0;
    hipGetDeviceCount(&numOfGPUs);

    auto totalGPUBlocks = (nFlagBlocks - 1) / GPUBlockSize + 1;
    auto numOfKernels = std::min(totalGPUBlocks / numOfGPUs, numOfGPUs);
    auto gpuBlocksPerKernel = (totalGPUBlocks - 1) / numOfKernels + 1;
    auto dataBlocksPerKernel = gpuBlocksPerKernel * GPUBlockSize;
    auto alignedKernelSize = dataBlocksPerKernel * DataBlockSize;
    
    // Allocate ----------------------------------
    printf("Allocating device variables... ");
    timer.begin();

    deviceInBuf = new uint8_t*[numOfKernels];
    deviceOutBuf = new uint8_t*[numOfKernels];
    deviceFlagIn = new CompressFlagBlock*[numOfKernels];

    for (int i = 0; i < numOfKernels; ++i) {
        cudaCheckError(hipSetDevice(i), "Failed to set device");

        auto kernelOutSize = std::min(alignedKernelSize, outSize - i * alignedKernelSize);
        auto numOfDataBlock = std::min(dataBlocksPerKernel, nFlagBlocks - i * dataBlocksPerKernel);

        cudaCheckError(hipMalloc((void**)&deviceInBuf[i], inSize), "Failed to allocate deviceInBuf");
        cudaCheckError(hipMalloc((void**)&deviceOutBuf[i], kernelOutSize), "Failed to allocate deviceOutBuf");
        cudaCheckError(hipMalloc((void**)&deviceFlagIn[i], sizeof(CompressFlagBlock) * numOfDataBlock),
            "Failed to allocate deviceFlagIn");

        hipDeviceSynchronize();
    }
    printf("%.6fs\n", timer.end());

    // Copy: host to device -----------------------
    printf("Copying from host to device => GPU");
    timer.begin();

    for (int i = 0; i < numOfKernels; ++i) {
        cudaCheckError(hipSetDevice(i), "Failed to set device");

        auto numOfDataBlock = std::min(dataBlocksPerKernel, nFlagBlocks - i * dataBlocksPerKernel);

        printf(" [%d]", i);
        fflush(stdout);

        cudaCheckError(hipMemcpyAsync(deviceInBuf[i], inBuf, inSize, hipMemcpyHostToDevice),
            "Failed to copy inBuf to device");

        cudaCheckError(hipMemcpyAsync(deviceFlagIn[i],
                           flagIn + i * dataBlocksPerKernel, sizeof(CompressFlagBlock) * numOfDataBlock,
                           hipMemcpyHostToDevice),
            "Failed to copy flagIn to device");
    }

    for (int i = 0; i < numOfKernels; ++i) {
        cudaCheckError(hipSetDevice(i), "Failed to set device");
        cudaCheckError(hipDeviceSynchronize(), "Failed to synchronize");
    }
    printf("... %.6fs\n", timer.end());

    // Launch kernel ----------------------------------
    printf("Launching kernel => GPU ");
    timer.begin();

    for (int i = 0; i < numOfKernels; ++i) {
        cudaCheckError(hipSetDevice(i), "Failed to set device");

        auto kernelInSize = std::min(alignedKernelSize, inSize - i * alignedKernelSize);
        auto numOfDataBlock = std::min(dataBlocksPerKernel, nFlagBlocks - i * dataBlocksPerKernel);
        auto numOfGPUBlock = std::min(gpuBlocksPerKernel, totalGPUBlocks - i * gpuBlocksPerKernel);

        printf(" [%d]", i);
        fflush(stdout);

        DecompressKernel<<<numOfGPUBlock, GPUBlockSize>>>(deviceFlagIn[i], numOfDataBlock, 
            deviceInBuf[i], deviceOutBuf[i]);
    }

    for (int i = 0; i < numOfKernels; ++i) {
        cudaCheckError(hipSetDevice(i), "Failed to set device");
        cudaCheckError(hipDeviceSynchronize(), "Failed to launch multi-GPU kernel");
    }
    auto elasped = timer.end();
    printf("... %.6fs\n", elasped);

    // Copy: device to host ---------------------------
    printf("Copying from device to host => GPU");
    timer.begin();

    for (int i = 0; i < numOfKernels; ++i) {
        cudaCheckError(hipSetDevice(i), "Failed to set device");

        auto kernelOutSize = std::min(alignedKernelSize, outSize - i * alignedKernelSize);

        printf(" [%d]", i);
        fflush(stdout);

        cudaCheckError(hipMemcpyAsync(outBuf + i * alignedKernelSize,
                           deviceOutBuf[i], kernelOutSize,
                           hipMemcpyDeviceToHost),
            "Failed to copy deviceOutBuf to host");
    }

    for (int i = 0; i < numOfKernels; ++i) {
        cudaCheckError(hipSetDevice(i), "Failed to set device");
        cudaCheckError(hipDeviceSynchronize(), "Failed to synchronize");
    }
    printf("... %.6fs\n", timer.end());

    for (int i = 0; i < numOfKernels; ++i) {
        cudaCheckError(hipSetDevice(i), "Failed to set device");
        hipFree(deviceInBuf[i]);
        hipFree(deviceOutBuf[i]);
        hipFree(deviceFlagIn[i]);
    }

    delete[] deviceInBuf;
    delete[] deviceOutBuf;
    delete[] deviceFlagIn;

    return std::make_pair(true, -1);
}
