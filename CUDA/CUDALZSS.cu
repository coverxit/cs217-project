#include "hip/hip_runtime.h"
#include <utility>

#include <memory.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "../Settings.h"

#include "../BitHelper.h"
#include "../LZSSInterface.h"
#include "../TimerHelper.hpp"

#include "../MatchHelper/MatchHelper.h"
#include "CUDALZSS.h"

__global__ void CompressKernel(const uint8_t* deviceInBuf, int inSize,
    uint8_t* deviceOutBuf, int* deviceOutSize,
    CompressFlagBlock* deviceFlagOut, int nFlagBlocks, int* deviceFlagSize,
    int* deviceNumBlocksDone);

#define cudaCheckError(op, msg)    \
    do {                           \
        hipError_t ret = (op);    \
        if ((ret) != hipSuccess)  \
            _gerror((ret), (msg)); \
    } while (false)

inline void _gerror(hipError_t hipError_t, const char* msg)
{
    fprintf(stderr, "%s, hipError_t = %d\n", msg, hipError_t);
    exit(-1);
}

std::pair<bool, double> CUDALZSS::compress(const uint8_t* inBuf, int inSize,
    uint8_t* outBuf, int& outSize,
    CompressFlagBlock* flagOut, int nFlagBlocks, int& flagSize)
{
    Timer timer(false);

    uint8_t *deviceInBuf, *deviceOutBuf;
    CompressFlagBlock* deviceFlagOut;
    int *deviceOutSize, *deviceFlagSize, *deviceNumBlocksDone;

    // Allocate ----------------------------------
    printf("Allocating device variables... ");
    fflush(stdout);

    timer.begin();
    cudaCheckError(hipMalloc((void**)&deviceInBuf, inSize), "Failed to allocate deviceInBuf");

    cudaCheckError(hipMalloc((void**)&deviceOutBuf, outSize), "Failed to allocate deviceOutBuf");
    cudaCheckError(hipMalloc((void**)&deviceOutSize, sizeof(int)), "Failed to allocate deviceOutSize");

    cudaCheckError(hipMalloc((void**)&deviceFlagOut, sizeof(CompressFlagBlock) * nFlagBlocks),
        "Failed to allocate deviceFlagOut");
    cudaCheckError(hipMalloc((void**)&deviceFlagSize, sizeof(int)), "Failed to allocate deviceFlagSize");
    cudaCheckError(hipMalloc((void**)&deviceNumBlocksDone, sizeof(int)), "Failed to allocate deviceNumBlocksDone");
    hipDeviceSynchronize();
    printf("%.6f s\n", timer.end());

    // Copy: host to device -----------------------
    printf("Copying data from host to device... ");
    fflush(stdout);

    timer.begin();
    cudaCheckError(hipMemcpy(deviceInBuf, inBuf, inSize, hipMemcpyHostToDevice),
        "Failed to copy deviceInBuf to device");
    cudaCheckError(hipMemset(deviceFlagOut, 0, sizeof(CompressFlagBlock) * nFlagBlocks),
        "Failed to set deviceFlagOut to 0");
    hipDeviceSynchronize();
    printf("%.6f s\n", timer.end());

    // Launch kernel ------------------------------
    printf("Launching kernel... ");
    fflush(stdout);

    timer.begin();
    CompressKernel<<<nFlagBlocks, GPUBlockSize>>>(deviceInBuf, inSize,
        deviceOutBuf, deviceOutSize,
        deviceFlagOut, nFlagBlocks, deviceFlagSize,
        deviceNumBlocksDone);
    auto elapsed = timer.end();
    cudaCheckError(hipDeviceSynchronize(), "Failed to launch kernel");
    printf("%.6f s\n", elapsed);

    // Copy: device to host -----------------------
    printf("Copying data from device to host... ");
    fflush(stdout);

    timer.begin();
    cudaCheckError(hipMemcpy(&outSize, deviceOutSize, sizeof(int), hipMemcpyDeviceToHost),
        "Failed to copy deviceOutSize to host");
    cudaCheckError(hipMemcpy(&flagSize, deviceFlagSize, sizeof(int), hipMemcpyDeviceToHost),
        "Failed to copy deviceFlagSize to host");

    cudaCheckError(hipMemcpy(outBuf, deviceOutBuf, inSize, hipMemcpyDeviceToHost),
        "Failed to copy deviceOutBuf to host");
    cudaCheckError(hipMemcpy(flagOut, deviceFlagOut, sizeof(CompressFlagBlock) * nFlagBlocks, hipMemcpyDeviceToHost),
        "Failed to copy deviceFlagOut to host");
    hipDeviceSynchronize();
    printf("%.6f s\n", elapsed);
    fflush(stdout);

    hipFree(deviceInBuf);
    hipFree(deviceOutBuf);
    hipFree(deviceOutSize);
    hipFree(deviceFlagOut);
    hipFree(deviceFlagSize);
    hipFree(deviceNumBlocksDone);

    return std::make_pair(true, elapsed);
}

std::pair<bool, double> CUDALZSS::decompress(CompressFlagBlock* flagIn, int nFlagBlocks,
    const uint8_t* inBuf, int inSize, uint8_t* outBuf)
{
    Timer timer(false);

    timer.begin();
    //DecompressKernel<<<,>>>();
    auto elapsed = timer.end();

    return std::make_pair(true, elapsed);
}
