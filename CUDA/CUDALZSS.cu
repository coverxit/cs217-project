#include "hip/hip_runtime.h"
#include <utility>

#include <memory.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "../Settings.h"

#include "../LZSSInterface.h"
#include "../TimerHelper.hpp"

#include "../MatchHelper/MatchHelper.h"
#include "CUDALZSS.h"

#define cudaCheckError(op, msg) \
    do { hipError_t ret = (op); if ((ret) != hipSuccess) _gerror((ret), (msg), __LINE__); } while (false)

inline void _gerror(hipError_t hipError_t, const char* msg, int lineNo) {
    fprintf(stderr, "[%s:%d] %s, hipError_t = %d\n", lineNo, __FILE__, msg, hipError_t);
    exit(-1);
}

__global__ void CompressKernel(const uint8_t* deviceInBuf, int inSize,
    uint8_t* deviceOutBuf, int* deviceOutSize, CompressFlagBlock* deviceFlagOut, int nFlagBlocks, int* deviceFlagSize) {
    
}

std::pair<bool, double> CUDALZSS::compress(const uint8_t* inBuf, int inSize,
    uint8_t* outBuf, int& outSize,
    CompressFlagBlock* flagOut, int nFlagBlocks, int& flagSize)
{
    Timer timer(false);

    uint8_t *deviceInBuf, *deviceOutBuf;
    CompressFlagBlock *deviceFlagOut;
    int *deviceOutSize, *deviceFlagSize;

    printf("Allocating device variables...\n");
    cudaCheckError(hipMalloc((void**) &deviceInBuf, inSize), "Failed to allocate device memory");
    cudaCheckError(hipMalloc((void**) &deviceOutBuf, outSize), "Failed to allocate device memory");
    cudaCheckError(hipMalloc((void**) &deviceOutSize, sizeof(int)), "Failed to allocate device memory");
    cudaCheckError(hipMalloc((void**) &deviceFlagSize, sizeof(int)), "Failed to allocate device memory");

    printf("Copying data from host to device...\n");
    cudaCheckError(hipMemcpy(deviceInBuf, inBuf, inSize, hipMemcpyHostToDevice), "Failed to copy memory to device");
    cudaCheckError(hipMemset(deviceFlagOut, 0, sizeof(CompressFlagBlock) * nFlagBlocks),
        "Failed to set device memory");

    timer.begin();
    //CompressKernel<<<,>>>();
    auto elapsed = timer.end();

    printf("Copying data from device to host...\n");
    cudaCheckError(hipMemcpy(&outSize, deviceOutSize, sizeof(int), hipMemcpyDeviceToHost),
        "Failed to copy memory to host");
    cudaCheckError(hipMemcpy(&flagSize, deviceFlagSize, sizeof(int), hipMemcpyDeviceToHost),
        "Failed to copy memory to host");

    cudaCheckError(hipMemcpy(outBuf, deviceOutBuf, outSize, hipMemcpyDeviceToHost), "Failed to copy memory to host");
    cudaCheckError(hipMemcpy(flagOut, deviceFlagOut, sizeof(CompressFlagBlock) * nFlagBlocks, hipMemcpyDeviceToHost),
        "Failed to copy memory to host");
    
    return std::make_pair(true, elapsed);
}

std::pair<bool, double> CUDALZSS::decompress(CompressFlagBlock* flagIn, int nFlagBlocks,
    const uint8_t* inBuf, int inSize, uint8_t* outBuf)
{
    Timer timer(false);

    timer.begin();
    //DecompressKernel<<<,>>>();
    auto elapsed = timer.end();

    return std::make_pair(true, elapsed);
}
