#include "hip/hip_runtime.h"
#include <algorithm>
#include <utility>

#include <memory.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "../Settings.h"

#include "../BitHelper.h"
#include "../LZSSInterface.h"
#include "../TimerHelper.hpp"

#include "../MatchHelper/MatchHelper.h"
#include "CUDALZSS.h"
#include "Kernel.h"

#define cudaCheckError(op, msg)    \
    do {                           \
        hipError_t ret = (op);    \
        if ((ret) != hipSuccess)  \
            _gerror((ret), (msg)); \
    } while (false)

inline void _gerror(hipError_t hipError_t, const char* msg)
{
    fprintf(stderr, "%s, CUDA Error (%d): %s\n", msg, hipError_t, hipGetErrorString(hipError_t));
    exit(-1);
}

std::pair<bool, double> CUDALZSS::compress(const uint8_t* inBuf, int inSize,
    uint8_t* outBuf, int& outSize,
    CompressFlagBlock* flagOut, int nFlagBlocks, int& flagSize)
{
    Timer timer(false);

    uint8_t *deviceInBuf, *deviceOutBuf;
    CompressFlagBlock* deviceFlagOut;
    int *deviceOutSize, *deviceFlagSize;
    int *hostOutSize, *hostFlagSize;
    hipStream_t *cudaStreams;

    auto numOfStreams = std::min(nFlagBlocks, NumOfCUDAStream);
    auto blocksPerStream = (nFlagBlocks - 1) / numOfStreams + 1;
    auto alignedStreamSize = blocksPerStream * DataBlockSize;

    // Create stream -----------------------------
    cudaStreams = new hipStream_t[numOfStreams];
    for (int i = 0; i < numOfStreams; ++i) {
        cudaCheckError(hipStreamCreate(&cudaStreams[i]), "Failed to create CUDA streams");
    }

    // Allocate ----------------------------------
    printf("Allocating device variables... ");
    fflush(stdout);

    timer.begin();
    cudaCheckError(hipMalloc((void**)&deviceInBuf, inSize), "Failed to allocate deviceInBuf");

    hostOutSize = new int[numOfStreams];
    hostFlagSize = new int[numOfStreams];
    cudaCheckError(hipMalloc((void**)&deviceOutBuf, inSize), "Failed to allocate deviceOutBuf");
    cudaCheckError(hipMalloc((void**)&deviceOutSize, numOfStreams * sizeof(int)), "Failed to allocate deviceOutSize");

    cudaCheckError(hipMalloc((void**)&deviceFlagOut, sizeof(CompressFlagBlock) * nFlagBlocks),
        "Failed to allocate deviceFlagOut");
    cudaCheckError(hipMalloc((void**)&deviceFlagSize, numOfStreams * sizeof(int)), "Failed to allocate deviceFlagSize");
    hipDeviceSynchronize();
    printf("%.6fs\n", timer.end());

    // Copy: host to device -----------------------
    printf("Copying data from host to device... ");
    fflush(stdout);

    timer.begin();

    for (int i = 0; i < numOfStreams; ++i) {
        auto streamSize = std::min(alignedStreamSize, inSize - i * alignedStreamSize);
        auto numOfBlock = std::min(blocksPerStream, nFlagBlocks - i * blocksPerStream);

        cudaCheckError(hipMemcpyAsync(deviceInBuf + i * alignedStreamSize, 
            inBuf + i * alignedStreamSize, streamSize, 
            hipMemcpyHostToDevice, cudaStreams[i]), 
            "Failed to copy inBuf to device");

        cudaCheckError(hipMemsetAsync(deviceFlagOut + i * blocksPerStream, 0, 
            sizeof(CompressFlagBlock) * numOfBlock, cudaStreams[i]),
            "Failed to set deviceFlagOut to 0");
    }

    cudaCheckError(hipMemset(deviceOutSize, 0, sizeof(int) * numOfStreams), "Failed to set deviceOutSize to 0");
    cudaCheckError(hipMemset(deviceFlagSize, 0, sizeof(int) * numOfStreams), "Failed to set deviceFlagSize to 0");
    hipDeviceSynchronize();
    printf("%.6fs\n", timer.end());

    // Launch kernel ------------------------------
    printf("Launching kernel... ");
    fflush(stdout);

    timer.begin();

    for (int i = 0; i < numOfStreams; ++i) {
        auto streamSize = std::min(alignedStreamSize, inSize - i * alignedStreamSize);
        auto numOfBlock = std::min(blocksPerStream, nFlagBlocks - i * blocksPerStream);

        CompressKernel<<<numOfBlock, GPUBlockSize, 0, cudaStreams[i]>>>(
            deviceInBuf + i * alignedStreamSize, streamSize,
            deviceOutBuf + i * alignedStreamSize, &deviceOutSize[i],
            deviceFlagOut + i * blocksPerStream, &deviceFlagSize[i]);
    }

    cudaCheckError(hipDeviceSynchronize(), "Failed to launch kernel");
    auto elapsed = timer.end();
    printf("%.6fs\n", elapsed);

    // Copy: device to host -----------------------
    printf("Copying data from device to host... ");
    fflush(stdout);

    timer.begin();
    
    for (int i = 0; i < numOfStreams; ++i) {
        auto streamSize = std::min(alignedStreamSize, inSize - i * alignedStreamSize);
        auto numOfBlock = std::min(blocksPerStream, nFlagBlocks - i * blocksPerStream);

        cudaCheckError(hipMemcpyAsync(outBuf + i * alignedStreamSize, 
            deviceOutBuf + i * alignedStreamSize, streamSize, 
            hipMemcpyDeviceToHost, cudaStreams[i]), 
            "Failed to copy deviceOutBuf to host");

        cudaCheckError(hipMemcpyAsync(flagOut + i * blocksPerStream,
            deviceFlagOut + i * blocksPerStream, numOfBlock * sizeof(CompressFlagBlock),
            hipMemcpyDeviceToHost, cudaStreams[i]),
            "Failed to copy deviceFlagOut to host");
    }

    cudaCheckError(hipMemcpy(hostOutSize, deviceOutSize, numOfStreams * sizeof(int), hipMemcpyDeviceToHost),
        "Failed to copy deviceOutSize to host");
    cudaCheckError(hipMemcpy(hostFlagSize, deviceFlagSize, numOfStreams * sizeof(int), hipMemcpyDeviceToHost),
        "Failed to copy deviceFlagSize to host");
    hipDeviceSynchronize();
    printf("%.6fs\n", timer.end());
    fflush(stdout);

    // Post process ------------------------------
    outSize = flagSize = 0;
    for (int i = 0; i < numOfStreams; ++i) {
        outSize += hostOutSize[i];
        flagSize += hostFlagSize[i];
    }

    // Cleanup -----------------------------------
    for (int i = 0; i < numOfStreams; ++i) {
        cudaCheckError(hipStreamDestroy(cudaStreams[i]), "Failed to destroy CUDA streams");
    }

    delete[] cudaStreams;
    delete[] hostOutSize;
    delete[] hostFlagSize;

    hipFree(deviceInBuf);
    hipFree(deviceOutBuf);
    hipFree(deviceOutSize);
    hipFree(deviceFlagOut);
    hipFree(deviceFlagSize);

    return std::make_pair(true, elapsed);
}

std::pair<bool, double> CUDALZSS::decompress(CompressFlagBlock* flagIn, int nFlagBlocks,
    const uint8_t* inBuf, int inSize, uint8_t* outBuf, int outSize)
{
    Timer timer(false);

    uint8_t *deviceInBuf, *deviceOutBuf;
    CompressFlagBlock* deviceFlagIn;

    // Allocate ----------------------------------
    printf("Allocating device variables... ");
    fflush(stdout);

    timer.begin();
    cudaCheckError(hipMalloc((void**)&deviceInBuf, inSize), "Failed to allocate deviceInBuf");
    cudaCheckError(hipMalloc((void**)&deviceOutBuf, outSize), "Failed to allocate deviceOutBuf");
    cudaCheckError(hipMalloc((void**)&deviceFlagIn, sizeof(CompressFlagBlock) * nFlagBlocks),
        "Failed to allocate deviceFlagIn");
    hipDeviceSynchronize();
    printf("%.6fs\n", timer.end());

    // Copy: host to device -----------------------
    printf("Copying data from host to device... ");
    fflush(stdout);

    timer.begin();
    cudaCheckError(hipMemcpy(deviceInBuf, inBuf, inSize, hipMemcpyHostToDevice),
        "Failed to copy inBuf to device");
    cudaCheckError(hipMemcpy(deviceFlagIn, flagIn, sizeof(CompressFlagBlock) * nFlagBlocks, hipMemcpyHostToDevice),
        "Failed to copy flagIn to device");
    hipDeviceSynchronize();
    printf("%.6fs\n", timer.end());

    // Launch kernel ------------------------------
    printf("Launching kernel... ");
    fflush(stdout);

    timer.begin();
    auto dimGrid = (nFlagBlocks - 1) / GPUBlockSize + 1;
    DecompressKernel<<<dimGrid, GPUBlockSize>>>(deviceFlagIn, nFlagBlocks, deviceInBuf, deviceOutBuf);
    cudaCheckError(hipDeviceSynchronize(), "Failed to launch kernel");
    auto elapsed = timer.end();
    printf("%.6fs\n", elapsed);

    // Copy: device to host -----------------------
    printf("Copying data from device to host... ");
    fflush(stdout);

    timer.begin();
    cudaCheckError(hipMemcpy(outBuf, deviceOutBuf, outSize, hipMemcpyDeviceToHost),
        "Failed to copy deviceOutBuf to host");
    hipDeviceSynchronize();
    printf("%.6fs\n", timer.end());
    fflush(stdout);

    hipFree(deviceInBuf);
    hipFree(deviceOutBuf);
    hipFree(deviceFlagIn);

    return std::make_pair(true, elapsed);
}
