#include "hip/hip_runtime.h"
#include <algorithm>
#include <utility>

#include <memory.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "../Settings.h"

#include "../BitHelper.h"
#include "../LZSSInterface.h"
#include "../TimerHelper.hpp"

#include "../MatchHelper/MatchHelper.h"
#include "CUDALZSS.h"
#include "Kernel.h"

#define cudaCheckError(op, msg)    \
    do {                           \
        hipError_t ret = (op);    \
        if ((ret) != hipSuccess)  \
            _gerror((ret), (msg)); \
    } while (false)

inline void _gerror(hipError_t hipError_t, const char* msg)
{
    fprintf(stderr, "%s, CUDA Error (%d): %s\n", msg, hipError_t, hipGetErrorString(hipError_t));
    exit(-1);
}

std::pair<bool, double> CUDALZSS::compress(const uint8_t* inBuf, int inSize,
    uint8_t* outBuf, int& outSize,
    CompressFlagBlock* flagOut, int nFlagBlocks, int& flagSize)
{
    Timer timer(false), timerKernel;

    uint8_t **deviceInBuf, **deviceOutBuf;
    CompressFlagBlock** deviceFlagOut;
    int **deviceOutSize, **deviceFlagSize;
    int *hostOutSize, *hostFlagSize;

    int numOfGPUs = 0;
    hipGetDeviceCount(&numOfGPUs);

    deviceOutSize = new int*[numOfKernels];
    deviceFlagSize = new int*[numOfKernels];
    hostOutSize = new int[numOfKernels];
    hostFlagSize = new int[numOfKernels];

    auto numOfKernels = std::min(nFlagBlocks / numOfGPUs, numOfGPUs);
    auto blocksPerKernel = (nFlagBlocks - 1) / numOfKernels + 1;
    auto alignedKernelSize = blocksPerKernel * DataBlockSize;

    // Allocate ----------------------------------
    printf("Allocating device variables... ");
    timer.begin();

    for (int i = 0; i < numOfKernels; ++i) {
        hipSetDevice(i);

        auto kernelSize = std::min(alignedKernelSize, inSize - i * alignedKernelSize);
        auto numOfBlock = std::min(blocksPerKernel, nFlagBlocks - i * blocksPerKernel);
        
        cudaCheckError(hipMalloc((void**)&deviceInBuf[i], kernelSize), "Failed to allocate deviceInBuf");

        cudaCheckError(hipMalloc((void**)&deviceOutBuf[i], kernelSize), "Failed to allocate deviceOutBuf");
        cudaCheckError(hipMalloc((void**)&deviceOutSize[i], sizeof(int)), "Failed to allocate deviceOutSize");

        cudaCheckError(hipMalloc((void**)&deviceFlagOut[i], sizeof(CompressFlagBlock) * numOfBlock),
            "Failed to allocate deviceFlagOut");
        cudaCheckError(hipMalloc((void**)&deviceFlagSize[i], sizeof(int)), "Failed to allocate deviceFlagSize");

        hipDeviceSynchronize();
    }
    printf("%.6fs\n", timer.end());

    // Multi-GPU ----------------------------------
    printf("Launching kernels on multiple GPUs...\n  => GPU:");
    timer.begin();
    
    for (int i = 0; i < numOfKernels; ++i) {
        hipSetDevice(i);

        auto kernelSize = std::min(alignedKernelSize, inSize - i * alignedKernelSize);
        auto numOfBlock = std::min(blocksPerKernel, nFlagBlocks - i * blocksPerKernel);

        // Copy: host to device -------------------
        cudaCheckError(hipMemcpyAsync(deviceInBuf[i], 
            inBuf + i * alignedKernelSize, kernelSize, 
            hipMemcpyHostToDevice),
            "Failed to copy inBuf to device");

        cudaCheckError(hipMemsetAsync(deviceFlagOut[i], 0, sizeof(CompressFlagBlock) * numOfBlock),
            "Failed to set deviceFlagOut to 0");

        cudaCheckError(hipMemsetAsync(deviceOutSize[i], 0, sizeof(int)),  "Failed to set deviceOutSize to 0");
        cudaCheckError(hipMemsetAsync(deviceFlagSize[i], 0, sizeof(int)),  "Failed to set deviceFlagSize to 0");

        printf(" [%d]", i);
        fflush(stdout);

        // Launch kernel ------------------------------
        CompressKernel<<<numOfBlock, GPUBlockSize>>>(
            deviceInBuf[i], kernelSize,
            deviceOutBuf[i], deviceOutSize[i],
            deviceFlagOut[i], deviceFlagSize[i]);

        // Copy: device to host -----------------------
        cudaCheckError(hipMemcpyAsync(outBuf + i * alignedKernelSize, 
            deviceOutBuf[i], kernelSize, 
            hipMemcpyDeviceToHost),
            "Failed to copy deviceOutBuf to host");

        cudaCheckError(hipMemcpyAsync(flagOut + i * blocksPerKernel,
            deviceFlagOut[i], numOfBlock * sizeof(CompressFlagBlock),
            hipMemcpyDeviceToHost),
            "Failed to copy deviceFlagOut to host");
        
        cudaCheckError(hipMemcpyAsync(hostOutSize + i, deviceOutSize[i], 
            sizeof(int), hipMemcpyDeviceToHost),
            "Failed to copy deviceOutSize to host");
        cudaCheckError(hipMemcpyAsync(hostFlagSize + i, deviceFlagSize[i], 
            sizeof(int), hipMemcpyDeviceToHost),
            "Failed to copy deviceFlagSize to host");

    }

    printf("\nWaiting for kernel exeuction complete... ");
    for (int i = 0; i < numOfKernels; ++i) {
        hipSetDevice(i);
        cudaCheckError(hipDeviceSynchronize(), "Failed to launch multi-GPU kernel");
    }
    printf("%.6fs\n", timer.end());

    printf("Post processing and cleanup... ");
    timer.begin();

    // Post process ------------------------------
    outSize = flagSize = 0;
    for (int i = 0; i < numOfKernels; ++i) {
        outSize += hostOutSize[i];
        flagSize += hostFlagSize[i];
    }

    for (int i = 0; i < numOfKernels; ++i) {
        hipSetDevice(i);

        hipFree(deviceInBuf[i]);
        hipFree(deviceOutBuf[i]);
        hipFree(deviceOutSize[i]);
        hipFree(deviceFlagOut[i]);
        hipFree(deviceFlagSize[i]);
    }

    delete[] deviceOutSize;
    delete[] deviceFlagSize;
    delete[] hostOutSize;
    delete[] hostFlagSize;

    printf("%.6fs\n", timer.end());
    return std::make_pair(true, timerKernel.end());
}

std::pair<bool, double> CUDALZSS::decompress(CompressFlagBlock* flagIn, int nFlagBlocks,
    const uint8_t* inBuf, int inSize, uint8_t* outBuf, int outSize)
{
    Timer timer(false), timerKernel;

    uint8_t *deviceInBuf, *deviceOutBuf;
    CompressFlagBlock* deviceFlagIn;

    // Allocate ----------------------------------
    printf("Allocating device variables... ");
    fflush(stdout);

    timer.begin();
    cudaCheckError(hipMalloc((void**)&deviceInBuf, inSize), "Failed to allocate deviceInBuf");
    cudaCheckError(hipMalloc((void**)&deviceOutBuf, outSize), "Failed to allocate deviceOutBuf");
    cudaCheckError(hipMalloc((void**)&deviceFlagIn, sizeof(CompressFlagBlock) * nFlagBlocks),
        "Failed to allocate deviceFlagIn");
    hipDeviceSynchronize();
    printf("%.6fs\n", timer.end());

    // Copy: host to device -----------------------
    printf("Copying data from host to device... ");
    fflush(stdout);

    timer.begin();
    cudaCheckError(hipMemcpy(deviceInBuf, inBuf, inSize, hipMemcpyHostToDevice),
        "Failed to copy inBuf to device");
    cudaCheckError(hipMemcpy(deviceFlagIn, flagIn, sizeof(CompressFlagBlock) * nFlagBlocks, hipMemcpyHostToDevice),
        "Failed to copy flagIn to device");
    hipDeviceSynchronize();
    printf("%.6fs\n", timer.end());

    // Launch kernel ------------------------------
    printf("Launching kernel... ");
    fflush(stdout);

    timer.begin();
    auto dimGrid = (nFlagBlocks - 1) / GPUBlockSize + 1;
    DecompressKernel<<<dimGrid, GPUBlockSize>>>(deviceFlagIn, nFlagBlocks, deviceInBuf, deviceOutBuf);
    cudaCheckError(hipDeviceSynchronize(), "Failed to launch kernel");
    printf("%.6fs\n", timer.end());

    // Copy: device to host -----------------------
    printf("Copying data from device to host... ");
    fflush(stdout);

    timer.begin();
    cudaCheckError(hipMemcpy(outBuf, deviceOutBuf, outSize, hipMemcpyDeviceToHost),
        "Failed to copy deviceOutBuf to host");
    hipDeviceSynchronize();
    printf("%.6fs\n", timer.end());
    fflush(stdout);

    hipFree(deviceInBuf);
    hipFree(deviceOutBuf);
    hipFree(deviceFlagIn);

    return std::make_pair(true, timerKernel.end());
}
