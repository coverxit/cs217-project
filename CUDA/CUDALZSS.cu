#include <utility>

#include <memory.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "../Settings.h"

#include "../BitHelper.h"
#include "../LZSSInterface.h"
#include "../TimerHelper.hpp"

#include "../MatchHelper/MatchHelper.h"
#include "CUDALZSS.h"
#include "Kernel.h"

#define cudaCheckError(op, msg)    \
    do {                           \
        hipError_t ret = (op);    \
        if ((ret) != hipSuccess)  \
            _gerror((ret), (msg)); \
    } while (false)

inline void _gerror(hipError_t hipError_t, const char* msg)
{
    fprintf(stderr, "%s, hipError_t = %d\n", msg, hipError_t);
    exit(-1);
}

std::pair<bool, double> CUDALZSS::compress(const uint8_t* inBuf, int inSize,
    uint8_t* outBuf, int& outSize,
    CompressFlagBlock* flagOut, int nFlagBlocks, int& flagSize)
{
    Timer timer(false);

    uint8_t *deviceInBuf, *deviceOutBuf;
    CompressFlagBlock* deviceFlagOut;
    int *deviceOutSize, *deviceFlagSize;

    // Allocate ----------------------------------
    printf("Allocating device variables... ");
    fflush(stdout);

    timer.begin();
    cudaCheckError(hipMalloc((void**)&deviceInBuf, inSize), "Failed to allocate deviceInBuf");

    cudaCheckError(hipMalloc((void**)&deviceOutBuf, outSize), "Failed to allocate deviceOutBuf");
    cudaCheckError(hipMalloc((void**)&deviceOutSize, sizeof(int)), "Failed to allocate deviceOutSize");

    cudaCheckError(hipMalloc((void**)&deviceFlagOut, sizeof(CompressFlagBlock) * nFlagBlocks),
        "Failed to allocate deviceFlagOut");
    cudaCheckError(hipMalloc((void**)&deviceFlagSize, sizeof(int)), "Failed to allocate deviceFlagSize");
    hipDeviceSynchronize();
    printf("%.6fs\n", timer.end());

    // Copy: host to device -----------------------
    printf("Copying data from host to device... ");
    fflush(stdout);

    timer.begin();
    cudaCheckError(hipMemcpy(deviceInBuf, inBuf, inSize, hipMemcpyHostToDevice),
        "Failed to copy inBuf to device");
    cudaCheckError(hipMemset(deviceFlagOut, 0, sizeof(CompressFlagBlock) * nFlagBlocks),
        "Failed to set deviceFlagOut to 0");
    hipDeviceSynchronize();
    printf("%.6fs\n", timer.end());

    // Launch kernel ------------------------------
    printf("Launching kernel...\n");
    fflush(stdout);

    timer.begin();
    CompressKernel<<<nFlagBlocks, GPUBlockSize>>>(deviceInBuf, inSize,
        deviceOutBuf, deviceOutSize,
        deviceFlagOut, nFlagBlocks, deviceFlagSize);
    cudaCheckError(hipDeviceSynchronize(), "Failed to launch kernel");
    auto elapsed = timer.end();

    // Copy: device to host -----------------------
    printf("Copying data from device to host... ");
    fflush(stdout);

    timer.begin();
    cudaCheckError(hipMemcpy(&outSize, deviceOutSize, sizeof(int), hipMemcpyDeviceToHost),
        "Failed to copy deviceOutSize to host");
    cudaCheckError(hipMemcpy(&flagSize, deviceFlagSize, sizeof(int), hipMemcpyDeviceToHost),
        "Failed to copy deviceFlagSize to host");

    cudaCheckError(hipMemcpy(outBuf, deviceOutBuf, inSize, hipMemcpyDeviceToHost),
        "Failed to copy deviceOutBuf to host");
    cudaCheckError(hipMemcpy(flagOut, deviceFlagOut, sizeof(CompressFlagBlock) * nFlagBlocks, hipMemcpyDeviceToHost),
        "Failed to copy deviceFlagOut to host");
    hipDeviceSynchronize();
    printf("%.6fs\n", timer.end());
    fflush(stdout);

    hipFree(deviceInBuf);
    hipFree(deviceOutBuf);
    hipFree(deviceOutSize);
    hipFree(deviceFlagOut);
    hipFree(deviceFlagSize);

    return std::make_pair(true, elapsed);
}

std::pair<bool, double> CUDALZSS::decompress(CompressFlagBlock* flagIn, int nFlagBlocks,
    const uint8_t* inBuf, int inSize, uint8_t* outBuf, int outSize)
{
    Timer timer(false);

    uint8_t *deviceInBuf, *deviceOutBuf, *deviceFlagIn;

    // Allocate ----------------------------------
    printf("Allocating device variables... ");
    fflush(stdout);

    timer.begin();
    cudaCheckError(hipMalloc((void**)&deviceInBuf, inSize), "Failed to allocate deviceInBuf");
    cudaCheckError(hipMalloc((void**)&deviceOutBuf, outSize), "Failed to allocate deviceOutBuf");
    cudaCheckError(hipMalloc((void**)&deviceFlagIn, sizeof(CompressFlagBlock) * nFlagBlocks),
        "Failed to allocate deviceFlagIn");
    hipDeviceSynchronize();
    printf("%.6fs\n", timer.end());

    // Copy: host to device -----------------------
    printf("Copying data from host to device... ");
    fflush(stdout);

    timer.begin();
    cudaCheckError(hipMemcpy(deviceInBuf, inBuf, inSize, hipMemcpyHostToDevice),
        "Failed to copy inBuf to device");
    cudaCheckError(hipMemcpy(deviceFlagIn, flagIn, sizeof(CompressFlagBlock) * nFlagBlocks, hipMemcpyHostToDevice),
        "Failed to copy flagIn to device");
    hipDeviceSynchronize();
    printf("%.6fs\n", timer.end());

    // Launch kernel ------------------------------
    printf("Launching kernel...\n");
    fflush(stdout);

    timer.begin();
    auto dimGrid = (nFlagBlocks - 1) / GPUBlockSize + 1;
    DecompressKernel<<<dimGrid, GPUBlockSize>>>(deviceFlagIn, nFlagBlocks, deviceInBuf, deviceOutBuf);
    cudaCheckError(hipDeviceSynchronize(), "Failed to launch kernel");
    auto elapsed = timer.end();

    // Copy: device to host -----------------------
    printf("Copying data from device to host... ");
    fflush(stdout);

    timer.begin();
    cudaCheckError(hipMemcpy(outBuf, deviceOutBuf, outSize, hipMemcpyDeviceToHost),
        "Failed to copy deviceOutBuf to host");
    hipDeviceSynchronize();
    printf("%.6fs\n", timer.end());
    fflush(stdout);

    hipFree(deviceInBuf);
    hipFree(deviceOutBuf);
    hipFree(deviceFlagIn);

    return std::make_pair(true, elapsed);
}
